#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 512

// Global function means it will be executed on the device (GPU)
__global__ void add(int *in1, int *in2, int *out)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	out[index] = in1[index] + in2[index];
}

void random_ints(int *i, int size)
{
	for(int k=0; k<size; k++)
	{
		i[k]=rand()%50;
	}
}

int *testmain(int num, int threads) 
{
	int *in1, *in2, *out; // host copies of inputs and output
	int *d_in1, *d_in2, *d_out; // device copies of inputs and output
	int size = num * sizeof(int);

	// Alloc space for device copies of three vectors
	hipMalloc((void **)&d_in1, size);
	hipMalloc((void **)&d_in2, size);
	hipMalloc((void **)&d_out, size);

	// Alloc space for host copies of the three vectors and setup input values
	in1 = (int *)malloc(size); random_ints(in1, num);
	in2 = (int *)malloc(size); random_ints(in2, num);
	out = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<num/threads,threads>>>(d_in1, d_in2, d_out);
	// Wait for the GPU to finish
	hipDeviceSynchronize();
	// Copy result back to host
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(in1); free(in2); free(out);
	hipFree(d_in1); hipFree(d_in2); hipFree(d_out);
	return out;
}
